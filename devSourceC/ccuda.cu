#include"globals.h"
void freeCudaPointer(float ** pointer) {
	hipError_t(hipHostFree(*pointer));
}
void allocCudaPointer(float ** pointer, size_t size) {
	hipError_t(hipHostMalloc((void**)pointer, size));
}
