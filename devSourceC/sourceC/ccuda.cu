#include"globals.h"
void freeCudaPointer(real ** pointer) {
	hipError_t(hipHostFree(*pointer));
}
void allocCudaPointer(real ** pointer, size_t size) {
	hipError_t(hipHostMalloc((void**)pointer, size));
}
