#include"globals.h"
#include"cudaGlobals.h"
#include"hipError_t.h"
float * devPii[2];
float * devTau[2];
float * devAfloat[2];
float * devAImag[2];
float * devBfloat[2];
float * devBImag[2];
float * devII[2];
int * devNmax[2];
hipStream_t stream[2];
float * devReferences[2];
float * devErr[2];
float * devPatterns[2];
float * devInvRSquare[2];
float * devPSquare[2];
hipStream_t streamRef[2];
float * devMin[2];
float * devMax[2];
int * devMinIndex[2];
float * devMedian[2];
int * devOut;


void freeCudaPointer(void ** pointer) {
	CudaSafeCall(hipHostFree(*pointer));
}
void allocCudaPointer(void ** pointer, size_t size) {
	CudaSafeCall(hipHostMalloc((void**)pointer, size));
}
	
void mallocCudaReferences(int i, int const mPatterns, int const nPatterns, int const mReferences, int const nReferences ) {
			CudaSafeCall(hipStreamCreate(&streamRef[i]));
			CudaSafeCall(hipMalloc((void**)&devPatterns[i], mPatterns*nPatterns*sizeof(float)));
			CudaSafeCall(hipMalloc((void**)&devReferences[i],mReferences*nReferences*sizeof(float)));
			CudaSafeCall(hipMalloc((void**)&devInvRSquare[i], mReferences*sizeof(float)));
			CudaSafeCall(hipMalloc((void**)&devPSquare[i], mPatterns*sizeof(float)));
			CudaSafeCall(hipMalloc((void**)&devErr[i], mPatterns*mReferences*sizeof(float)));
			CudaSafeCall(hipMalloc((void**)&devMin[i], mPatterns*sizeof(float)));
			CudaSafeCall(hipMalloc((void**)&devMax[i], mPatterns*sizeof(float))); //TODO: czy rozmiar dobry? (04.04.13 by szmigacz)
			CudaSafeCall(hipMalloc((void**)&devMinIndex[i], mPatterns*sizeof(int)));
			CudaSafeCall(hipMalloc((void**)&devMedian[i], mPatterns*sizeof(float)));

}

void freeCudaMemory() {
	#ifdef CUDA
		for(int i=0;i<2;i++) {
			CudaSafeCall(hipStreamSynchronize(stream[i]));
		}
	
		for(int i=0;i<2;i++) {
			CudaSafeCall(hipFree(devPii[i]));
			CudaSafeCall(hipFree(devTau[i]));
			CudaSafeCall(hipFree(devAfloat[i]));
			CudaSafeCall(hipFree(devAImag[i]));
			CudaSafeCall(hipFree(devBfloat[i]));
			CudaSafeCall(hipFree(devBImag[i]));
			CudaSafeCall(hipFree(devII[i]));
			CudaSafeCall(hipFree(devNmax[i]));
			CudaSafeCall(hipStreamDestroy(stream[i]));
		}
	#endif //CUDA
}

void freeCudaRefMemory() {
	#ifdef CUDA
		for(int i=0;i<2;i++) {
			CudaSafeCall(hipStreamSynchronize(streamRef[i]));
		}
		for(int i=0;i<2;i++) {
	
			CudaSafeCall(hipFree(devInvRSquare[i]));
			CudaSafeCall(hipFree(devPSquare[i]));
			CudaSafeCall(hipFree(devErr[i]));
			CudaSafeCall(hipFree(devPatterns[i]));
			CudaSafeCall(hipFree(devReferences[i]));
			CudaSafeCall(hipFree(devMin[i]));
			CudaSafeCall(hipFree(devMax[i]));
			CudaSafeCall(hipFree(devMinIndex[i]));
			CudaSafeCall(hipFree(devMedian[i]));
			CudaSafeCall(hipStreamDestroy(streamRef[i]));
		}
	#endif //CUDA
}

void cudaFinalize() {
	#ifdef CUDA
		CudaSafeCall(hipDeviceSynchronize());
	#endif //CUDA
}
void cuda1stPolarizationSync() {
	#ifdef CUDA
		CudaSafeCall(hipStreamSynchronize(streamRef[0]));
		CudaSafeCall(hipStreamSynchronize(streamRef[1]));
	#endif //CUDA
}

void freeCudaMemoryMin() {
	#ifdef CUDA
			CudaSafeCall(hipFree(devOut));
	#endif //CUDA
}
